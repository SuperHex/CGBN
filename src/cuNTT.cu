#include "hip/hip_runtime.h"
#include <string>
#include <exception>
#include <cassert>

#include <thrust/gather.h>
#include <thrust/device_ptr.h>

#include <cuNTT/cuNTT.hpp>
#include <cuNTT/common.cuh>
#include <cuNTT/radix2.cuh>
#include <cuNTT/radix4.cuh>
#include <cuNTT/arithmetic.cuh>

struct cuda_runtime_error : std::runtime_error {
    using std::runtime_error::runtime_error;
};

constexpr size_t calc_blocks(size_t len) { return (len + cuNTT_IPB - 1) / cuNTT_IPB; }

void cuda_check(hipError_t status, const char *action, const char *file, int line) {
    std::string err;
    if(status!=hipSuccess) {
        err += hipGetErrorString(status);
        // printf("CUDA error occurred: %s\n", hipGetErrorString(status));
        if(action!=NULL) {
            err += " while running ";
            err += action;
            err += "  (";
            err += file;
            err += ", line ";
            err += line;
            err += ") \n";
            // printf("While running %s   (file %s, line %d)\n", action, file, line);
        }
        throw cuda_runtime_error(err);
    }
}

void cgbn_check(cgbn_error_report_t *report, const char *file, int32_t line) {
    // check for cgbn errors
    if(cgbn_error_report_check(report)) {
        printf("\n");
        printf("CGBN error occurred: %s\n", cgbn_error_string(report));

        if(report->_instance!=0xFFFFFFFF) {
            printf("Error reported by instance %d", report->_instance);
            if(report->_blockIdx.x!=0xFFFFFFFF || report->_threadIdx.x!=0xFFFFFFFF)
                printf(", ");
            if(report->_blockIdx.x!=0xFFFFFFFF)
                printf("blockIdx=(%d, %d, %d) ", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
            if(report->_threadIdx.x!=0xFFFFFFFF)
                printf("threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
            printf("\n");
        }
        else {
            printf("Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
            printf("threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
        }
        if(file!=NULL)
            printf("file %s, line %d\n", file, line);
        throw cuda_runtime_error("CGBN error");
    }
}

#include <cgbn/cgbn.cu>

namespace cuNTT {

void to_mpz(mpz_class& r, const device_mem_t &x, uint32_t count) {
    mpz_import(r.get_mpz_t(), count, -1, sizeof(uint32_t), 0, 0, x._limbs);
}

void from_mpz(device_mem_t &x, const mpz_class& s, uint32_t count) {
    assert(mpz_sizeinbase(s.get_mpz_t(), 2) <= count*32);
    
    size_t words;
    mpz_export(x._limbs, &words, -1, sizeof(uint32_t), 0, 0, s.get_mpz_t());
    while(words < count)
        x._limbs[words++] = 0;
}

void permute_bit_reversal_radix2(device_mem_t *x, int N, int log2N) {
    kernel::permute_bit_reversal<2><<<calc_blocks(N), cuNTT_TPB>>>(x, x, N, log2N);
}

void permute_bit_reversal_radix4(device_mem_t *x, int N, int log2N) {
    kernel::permute_bit_reversal<4><<<calc_blocks(N), cuNTT_TPB>>>(x, x, N, log2N);
}

void EltwiseAddMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   const device_mem_t * const __restrict__ y,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseAddMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, y, N, modulus);
}

void EltwiseAddMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   device_mem_t scalar,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseAddMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, scalar, N, modulus);
}

void EltwiseSubMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   const device_mem_t * const __restrict__ y,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseSubMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, y, N, modulus);
}

void EltwiseSubMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   device_mem_t scalar,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseSubMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, scalar, N, modulus);
}

void EltwiseSubMod(device_mem_t *out,
                   device_mem_t scalar,
                   const device_mem_t * const __restrict__ x,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseSubMod<<<calc_blocks(N), cuNTT_TPB>>>(out, scalar, x, N, modulus);
}

void EltwiseMultMod(device_mem_t *out,
                    const device_mem_t * const __restrict__ x,
                    const device_mem_t * const __restrict__ y,
                    int N,
                    device_mem_t modulus)
{
    kernel::EltwiseMultMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, y, N, modulus);
}

void EltwiseMultMod(device_mem_t *out,
                    const device_mem_t * const __restrict__ x,
                    device_mem_t scalar,
                    int N,
                    device_mem_t modulus)
{
    kernel::EltwiseMultMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, scalar, N, modulus);
}

void EltwiseMontMultMod(device_mem_t *out,
                        const device_mem_t * const __restrict__ x,
                        device_mem_t adjusted_scalar,
                        int N)
{
    kernel::EltwiseMontMultMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, adjusted_scalar, N);
}

void EltwiseDivMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   const device_mem_t * const __restrict__ y,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseDivMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, y, N, modulus);
}

void EltwiseInvMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseInvMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, N, modulus);
}

void EltwiseFMAMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   device_mem_t scalar,
                   const device_mem_t * const __restrict__ y,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseFMAMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, scalar, y, N, modulus);
}

void EltwiseFMAMod(device_mem_t *out,
                   const device_mem_t * const __restrict__ x,
                   const device_mem_t * const __restrict__ r,
                   const device_mem_t * const __restrict__ y,
                   int N,
                   device_mem_t modulus)
{
    kernel::EltwiseFMAMod<<<calc_blocks(N), cuNTT_TPB>>>(out, x, r, y, N, modulus);
}

void UpdateQuadratic(device_mem_t *out,
                     const device_mem_t * const __restrict__ x,
                     const device_mem_t * const __restrict__ y,
                     const device_mem_t * const __restrict__ z,
                     device_mem_t r,
                     int N,
                     device_mem_t modulus)
{
    kernel::UpdateQuadratic<<<calc_blocks(N), cuNTT_TPB>>>(out, x, y, z, r, N, modulus);
}

void EltwiseBitDecompose(device_mem_t *out[],
                         const device_mem_t * const __restrict__ x,
                         int N,
                         int bits)
{
    kernel::EltwiseBitDecompose<<<calc_blocks(N), cuNTT_TPB>>>(out, x, N, bits);
}

void ntt_init_global(const mpz_class& p) {
    mpz_class two_p = p * 2, four_p = p * 4, eight_p = p * 8;
    device_mem_t device_p, device_2p, device_4p, device_8p;

    from_mpz(device_p,  p,       cuNTT_LIMBS);
    from_mpz(device_2p, two_p,   cuNTT_LIMBS);
    from_mpz(device_4p, four_p,  cuNTT_LIMBS);
    // from_mpz(device_8p, eight_p, cuNTT_LIMBS);

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_p),  &device_p,  sizeof(device_mem_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_2p), &device_2p, sizeof(device_mem_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_4p), &device_4p, sizeof(device_mem_t)));
    // CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_8p), &device_8p, sizeof(device_mem_t)));

    mpz_class J;
    device_mem_t device_J;
    mpz_class beta = mpz_class(1) << cuNTT_BITS;

    mpz_invert(J.get_mpz_t(), p.get_mpz_t(), beta.get_mpz_t());
    from_mpz(device_J, J, cuNTT_LIMBS);
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gpu_J), &device_J, sizeof(device_mem_t)));
}


ntt_context::ntt_context(const mpz_class& p, size_t N, const mpz_class& nth_root_of_unity) : degree_(N) {
    cgbn_error_report_t *err;
    device_mem_t *omega, *omega_inv;
    
    CUDA_CHECK(cgbn_error_report_alloc(&err));
    CUDA_CHECK(hipMalloc((void **)&omega, sizeof(device_mem_t) * N));
    CUDA_CHECK(hipMalloc((void **)&omega_inv, sizeof(device_mem_t) * N));

    err_ = std::unique_ptr<cgbn_error_report_t, cgbn_deleter>(err);
    device_omegas_ = std::unique_ptr<device_mem_t, device_deleter>(omega);
    device_omegas_inv_ = std::unique_ptr<device_mem_t, device_deleter>(omega_inv);

    device_mem_t w;
        
    // Precompute Forward NTT omegas
    from_mpz(w, nth_root_of_unity, cuNTT_LIMBS);
    kernel::precompute_omega_table<<<calc_blocks(N), cuNTT_TPB>>>(err_.get(), device_omegas_.get(), w, N);

    // Precompute Inverse NTT omegas
    mpz_class wi, Ni = N;
    mpz_invert(wi.get_mpz_t(), nth_root_of_unity.get_mpz_t(), p.get_mpz_t());
    mpz_invert(Ni.get_mpz_t(), Ni.get_mpz_t(), p.get_mpz_t());
    Ni = (Ni << cuNTT_BITS) % p;    // Adjust for Montgomery multiplication
        
    from_mpz(w, wi, cuNTT_LIMBS);
    from_mpz(N_inv_, Ni, cuNTT_LIMBS);
        
    kernel::precompute_omega_table<<<calc_blocks(N), cuNTT_TPB>>>(err_.get(), device_omegas_inv_.get(), w, N);
}

void ntt_context::ComputeForwardRadix2(device_mem_t *out, device_mem_t * const in) {
    size_t num_blocks = calc_blocks(degree_), num_parallel = 1;

    radix2_fft_forward(err_.get(), out, in,
                       device_omegas_.get(),
                       degree_,
                       dim3(num_blocks, num_parallel),
                       cuNTT_TPB);

}

// NOTE: uncomment 8p to use radix4!
// void ntt_context::ComputeForwardRadix4(device_mem_t *out, device_mem_t * const in) {
//     size_t num_blocks = calc_blocks(degree_), num_parallel = 1;
//     kernel::permute_bit_reversal<4><<<num_blocks, cuNTT_TPB>>> (out, in, degree_, std::log2(degree_));
//     radix4_fft_forward(err_.get(), out, in,
//                        device_omegas_.get(),
//                        degree_,
//                        dim3(num_blocks, num_parallel),
//                        cuNTT_TPB);
// }

void ntt_context::ComputeInverseRadix2(device_mem_t *out, device_mem_t * const in) {
    size_t num_blocks = calc_blocks(degree_), num_parallel = 1;
    radix2_fft_inverse(err_.get(), out, in,
                       device_omegas_inv_.get(),
                       N_inv_,
                       degree_,
                       dim3(num_blocks, num_parallel),
                       cuNTT_TPB);
}

// NOTE: uncomment 8p to use radix4!
// void ntt_context::ComputeInverseRadix4(device_mem_t *out, device_mem_t * const in) {
//     size_t num_blocks = calc_blocks(degree_), num_parallel = 1;
//     radix4_fft_inverse(err_.get(), out, in,
//                        device_omegas_inv_.get(),
//                        N_inv_,
//                        degree_,
//                        dim3(num_blocks, num_parallel),
//                        cuNTT_TPB);
//     kernel::permute_bit_reversal<4><<<num_blocks, cuNTT_TPB>>> (out, out, degree_, std::log2(degree_));
// }

__global__ void gatherKernel(const device_mem_t * __restrict__ input,
                             const size_t * __restrict__ indices,
                             device_mem_t * __restrict__ output,
                             size_t numIndices) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numIndices) {
        output[idx] = input[indices[idx]];
    }
}

void sample(device_mem_t *out, const device_mem_t *in, const size_t *index, size_t N)
{
    gatherKernel<<<calc_blocks(N), cuNTT_TPB>>>(in, index, out, N);
}


__global__ void memset_ui_kernel(device_mem_t arr[], int val, size_t N) {
    for (int idx = blockDim.x * blockIdx.x + threadIdx.x;
         idx < N;
         idx += blockDim.x * gridDim.x)
    {
        arr[idx]._limbs[0] = val;

        #pragma unroll 8
        for (int i = 1; i < cuNTT_LIMBS; i++) {
            arr[idx]._limbs[i] = 0;
        }
    }
}


__global__ void memset_mpz_kernel(device_mem_t arr[], device_mem_t val, size_t N) {
    for (int idx = blockDim.x * blockIdx.x + threadIdx.x;
         idx < N;
         idx += blockDim.x * gridDim.x)
    {
        arr[idx] = val;
    }
}

void memset_ui(device_mem_t arr[], int val, size_t N) {
    memset_ui_kernel<<<calc_blocks(N), cuNTT_TPB>>>(arr, val, N);
}

void memset_mpz(device_mem_t arr[], device_mem_t val, size_t N) {
    memset_mpz_kernel<<<calc_blocks(N), cuNTT_TPB>>>(arr, val, N);
}

}  // namespace cuNTT

